#include "hip/hip_runtime.h"
#include "kernel.cuh"

void Clear() {
#if defined _WIN32
    system("cls");
#elif defined (__LINUX__) || defined(__gnu_linux__) || defined(__linux__)
    system("clear");
#elif defined (__APPLE__)
    system("clear");
#endif
}

__global__ void lifeKernel(const ubyte *lifeData, uint worldWidth, uint worldHeight, ubyte *resultLifeData) {
    uint worldSize = worldWidth * worldHeight;

    for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x;
         cellId < worldSize;
         cellId += blockDim.x * gridDim.x) {

        uint x = cellId % worldWidth;
        uint yAbs = cellId - x;

        uint xLeft = (x + worldWidth - 1) % worldWidth;
        uint xRight = (x + 1) % worldWidth;

        uint yAbsUp = (yAbs + worldSize - worldWidth) % worldSize;
        uint yAbsDown = (yAbs + worldWidth) % worldSize;

        // Count alive cells.
        uint aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp] + lifeData[xRight + yAbsUp]
                          + lifeData[xLeft + yAbs] + lifeData[xRight + yAbs]
                          + lifeData[xLeft + yAbsDown] + lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

        resultLifeData[x + yAbs] = aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
    }
}

GpuLife::GpuLife() :
        d_data(nullptr),
        d_resultData(nullptr),
        m_worldWidth(0),
        m_worldHeight(0),
        m_worldSize(0),
        m_randomGen(std::random_device{}()) {
}

GpuLife::~GpuLife() {
    freeBuffers();
}

void GpuLife::freeBuffers() {
    HANDLE_ERROR(hipFree(d_data));
    d_data = nullptr;

    HANDLE_ERROR(hipFree(d_resultData));
    d_resultData = nullptr;
}

bool GpuLife::allocBuffers() {
    freeBuffers();
    m_worldSize = m_worldWidth * m_worldHeight;

    HANDLE_ERROR(hipMalloc(&d_data, m_worldSize));
    HANDLE_ERROR(hipMalloc(&d_resultData, m_worldSize));

    return true;
}

void GpuLife::resize(size_t width, size_t height) {
    freeBuffers();
    m_worldWidth = width;
    m_worldHeight = height;
}

void GpuLife::initRandom(ubyte *data, size_t size) {
    if (size != m_worldSize) {
        throw std::runtime_error("Size mismatch");
    }
    for (size_t i = 0; i < size; i++) {
        data[i] = m_randomGen() & 1;
    }
}

void GpuLife::copyToDevice(ubyte *data, size_t size) {
    if (size != m_worldSize) {
        throw std::runtime_error("Size mismatch");
    }
    HANDLE_ERROR(hipMemcpy(d_data, data, size, hipMemcpyHostToDevice));
}

void GpuLife::iterate(size_t iterations, size_t blockSize, int debug) {
    size_t gridSize = (m_worldWidth * m_worldHeight + blockSize - 1) / blockSize;

    for (size_t i = 0; i < iterations; i++) {
        lifeKernel<<<gridSize, blockSize>>>(d_data, m_worldWidth, m_worldHeight, d_resultData);
        HANDLE_ERROR(hipGetLastError());
        HANDLE_ERROR(hipDeviceSynchronize());
        std::swap(d_data, d_resultData);

        if (debug) {
            auto tmp = new ubyte[m_worldSize];
            HANDLE_ERROR(hipMemcpy(tmp, d_resultData, m_worldSize, hipMemcpyDeviceToHost));
            Clear();
            std::cout << "Iteration: " << i + 1 << std::endl;
            for (size_t k = 0; k < m_worldSize; k++) {
                std::cout << (tmp[k] ? "■" : "□");
                if ((k + 1) % m_worldWidth == 0) {
                    std::cout << std::endl;
                }
            }
        }
    }
}

void GpuLife::copyToHost(ubyte *data, size_t size) {
    if (size != m_worldSize) {
        throw std::runtime_error("Size mismatch");
    }
    HANDLE_ERROR(hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost));
}
